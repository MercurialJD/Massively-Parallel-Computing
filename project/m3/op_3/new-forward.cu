#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"


#define TILE_WIDTH 16
#define UNROLL_BATCH_SIZE 5000


__global__ void fusionedGEMMConv(const float *Kernel, const float *X, float *Y,
                                    const int Batch_start, const int Batch_end, const int Map_out, 
                                    const int Channel, const int Height, const int Width, const int K) {
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int W_unroll = Height_out * Width_out;        // Width of unrolled conv matrix

    const float *A = Kernel, *B = X;
    float *C = Y;
    const int numARows = Map_out;               const int numAColumns = Channel * K * K;
    const int numBRows = Channel * K * K;       const int numBColumns = Height_out * Width_out;
    const int numCRows = numARows;              const int numCColumns = numBColumns;

    #define A_2d(i1, i0) A[(i1) * numAColumns + (i0)]
    #define B_4d(i3, i2, i1, i0) B[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + (i0)]
    #define C_3d(i2, i1, i0) C[(i2) * (numCColumns * numCRows) + (i1) * (numCColumns) + (i0)]

    __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

    unsigned int bx = blockIdx.x; unsigned int by = blockIdx.y; unsigned int bz = blockIdx.z;
    unsigned int tx = threadIdx.x; unsigned int ty = threadIdx.y; unsigned int tz = threadIdx.z;

    unsigned int bat = bz * blockDim.z + tz;
    unsigned int row = by * blockDim.y + ty;
    unsigned int col = bx * blockDim.x + tx;

    float cVal = 0;
    for (size_t k = 0; k < (numAColumns - 1) / TILE_WIDTH + 1; ++k) {
        // Load partial A to shared memory
        if (row < numARows && (k * TILE_WIDTH + tx) < numAColumns)
            subTileA[ty][tx] = A_2d(row, k * TILE_WIDTH + tx);
        else
            subTileA[ty][tx] = 0.0f;

        // Load partial **UNROLLED** B to shared memory on the fly
        if ((k * TILE_WIDTH + ty) < numBRows && col < numBColumns) {
            int B_loc = (k * TILE_WIDTH + ty) * numBColumns + col;
            int row_u = B_loc / W_unroll, col_u = B_loc % W_unroll;

            int q = row_u % K;
            int p = (row_u / K) % K;
            int c = row_u / (K * K);

            int w = col_u % Width_out;
            int h = col_u / Width_out;

            subTileB[ty][tx] = B_4d(bat + Batch_start, c, h + p, w + q);
        } else {
            subTileB[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Sum per tile
        if (row < numCRows && col < numCColumns)
            for (size_t i = 0; i < TILE_WIDTH; ++i)
                cVal += subTileA[ty][i] * subTileB[i][tx];

        __syncthreads();
    }

    // Write to C if address is valid
    if (row < numCRows && col < numCColumns && bat < (Batch_end - Batch_start))
        C_3d(bat + Batch_start, row, col) = cVal;

    #undef A_2d
    #undef B_3d
    #undef C_3d
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    int input_size = Batch * Channel * Height * Width * sizeof(float);
    int output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float);
    int mask_size = Map_out * Channel * K * K * sizeof(float);

    hipMalloc((void **) device_input_ptr, input_size);
    hipMalloc((void **) device_output_ptr, output_size);
    hipMalloc((void **) device_mask_ptr, mask_size);

    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    // Partition input into smaller batches to avoid memory overflow
    for (int b = 0; b < Batch; b += UNROLL_BATCH_SIZE) {
        int curr_batch_size = min(Batch - b, UNROLL_BATCH_SIZE);
        dim3 gemmGrid( ceil(1.0 * (Height_out * Width_out) / TILE_WIDTH), \
                        ceil(1.0 * Map_out / TILE_WIDTH), curr_batch_size );
        dim3 gemmBlock(TILE_WIDTH, TILE_WIDTH);

        fusionedGEMMConv<<<gemmGrid, gemmBlock>>>(device_mask, device_input, device_output, \
                                        b, b + curr_batch_size, Map_out, Channel, Height, Width, K);
    }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float);
    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}